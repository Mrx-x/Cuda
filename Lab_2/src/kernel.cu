#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#define CUDA_CHECK_RETURN(value){\
	hipError_t _m_cudaStat = value;\
	if(_m_cudaStat != hipSuccess){\
		fprintf(stderr, "ERROR %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	}}

__global__ void add(float* a, float* b, float* c) {
	c[threadIdx.x + blockDim.x * blockIdx.x] = a[threadIdx.x + blockDim.x * blockIdx.x] + b[threadIdx.x + blockDim.x * blockIdx.x];
}

int main(void) {
	float* a, * b, * c;
	float* dev_a, * dev_b, * dev_c;
	int N = 1e6;
	float elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	int countDevice;
	hipGetDeviceCount(&countDevice);
	if (countDevice == 0) {
		fprintf(stderr, "[ERROR] - There is no device.\n");
	}
	else printf("Count device == [%d]\n", countDevice);

	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);	

	printf("Device name: %s.\n", deviceProp.name);
	printf("Total Global Memory = %lu bytes.\n", deviceProp.totalGlobalMem);
	printf("Total amount of constant memory: %lu bytes\n", deviceProp.totalConstMem);
	printf("Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
	printf("Total number of registers available per block: %d \n", deviceProp.regsPerBlock);
	printf("Warp size: %d\n", deviceProp.warpSize);
	printf("Max grid size: %lu.\n", deviceProp.maxGridSize);
	printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);

	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_a, N * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_c, N * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_b, N * sizeof(float)));

	a = (float*)calloc(N, sizeof(float));
	b = (float*)calloc(N, sizeof(float));
	c = (float*)calloc(N, sizeof(float));

	for (int i = 0; i < N; ++i)
	{
		a[i] = (float)rand() / (float)RAND_MAX;
		b[i] = (float)rand() / (float)RAND_MAX;
	}

	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);

	for (int t = 2; t <= 1024; t *= 2)
	{
		hipEventRecord(start, 0);
		add << < dim3(N / t), dim3(t) >> > (dev_a, dev_b, dev_c);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);

		fprintf(stderr, "gTest took %g per milliseconds\n", elapsedTime);
	}
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost));


	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}